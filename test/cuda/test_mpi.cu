#include "hip/hip_runtime.h"
#include <mpi.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <sstream>
#include <vector>
#include <random>

#define LINE_WIDTH 80
#define MAX_KERNELS 5

__global__ void kernel1(unsigned long rank, int call_id, bool enable_logging) {
    if (enable_logging) {
        printf("[KERNEL 1] GPU Block: %d | GPU Thread: %d | Call: %d | MPI Rank: %lu\n",
               blockIdx.x, threadIdx.x, call_id, rank);
    }
}

__global__ void kernel2(unsigned long rank, int call_id, bool enable_logging) {
    if (enable_logging) {
        printf("[KERNEL 2] GPU Block: %d | GPU Thread: %d | Call: %d | MPI Rank: %lu\n",
               blockIdx.x, threadIdx.x, call_id, rank);
    }
}

__global__ void kernel3(unsigned long rank, int call_id, bool enable_logging) {
    if (enable_logging) {
        printf("[KERNEL 3] GPU Block: %d | GPU Thread: %d | Call: %d | MPI Rank: %lu\n",
               blockIdx.x, threadIdx.x, call_id, rank);
    }
}

__global__ void kernel4(unsigned long rank, int call_id, bool enable_logging) {
    if (enable_logging) {
        printf("[KERNEL 4] GPU Block: %d | GPU Thread: %d | Call: %d | MPI Rank: %lu\n",
               blockIdx.x, threadIdx.x, call_id, rank);
    }
}

__global__ void kernel5(unsigned long rank, int call_id, bool enable_logging) {
    if (enable_logging) {
        printf("[KERNEL 5] GPU Block: %d | GPU Thread: %d | Call: %d | MPI Rank: %lu\n",
               blockIdx.x, threadIdx.x, call_id, rank);
    }
}

struct KernelConfig {
    int num_blocks;
    int num_threads;
    int num_calls;
    int kernel_number;
};

void print_line() {
    for (int i = 0; i < LINE_WIDTH; ++i) std::cout << "=";
    std::cout << "\n";
}

void run_kernel(int rank, const std::vector<KernelConfig>& configs, bool enable_logging) {
    if (enable_logging) {
        print_line();
        std::cout << "MPI Rank " << rank << " running on GPU " << 0 << "\n";
        print_line();
    }
    
    for (const auto& config : configs) {
        if (enable_logging) {
            std::cout << "Kernel " << config.kernel_number << " | Blocks: " << config.num_blocks
                      << " | Threads: " << config.num_threads << " | Calls: " << config.num_calls << "\n";
        }
        
        for (int j = 0; j < config.num_calls; ++j) {
            switch (config.kernel_number) {
                case 1:
                    kernel1<<<config.num_blocks, config.num_threads>>>((unsigned long)rank, j, enable_logging);
                    break;
                case 2:
                    kernel2<<<config.num_blocks, config.num_threads>>>((unsigned long)rank, j, enable_logging);
                    break;
                case 3:
                    kernel3<<<config.num_blocks, config.num_threads>>>((unsigned long)rank, j, enable_logging);
                    break;
                case 4:
                    kernel4<<<config.num_blocks, config.num_threads>>>((unsigned long)rank, j, enable_logging);
                    break;
                case 5:
                    kernel5<<<config.num_blocks, config.num_threads>>>((unsigned long)rank, j, enable_logging);
                    break;
                default:
                    return;
            }
            hipDeviceSynchronize();
            if (enable_logging) {
                std::cout << "Rank " << rank << " executed Kernel " << config.kernel_number 
                          << " | Blocks: " << config.num_blocks << " | Threads: " << config.num_threads
                          << " | Call ID: " << j << "\n";
            }
        }
    }
}

int main(int argc, char** argv) {
    MPI_Init(&argc, &argv);
    
    int rank, world_size;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &world_size);
    
    bool enable_logging = false;
    bool randomize = false;
    int num_blocks = 1, num_threads = 32, num_calls = 1, kernel_number = 1;
    
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<int> dist_blocks(1, 16);
    std::uniform_int_distribution<int> dist_threads(32, 512);
    std::uniform_int_distribution<int> dist_calls(1, 10);
    std::uniform_int_distribution<int> dist_kernel(1, 5);
    
    for (int i = 1; i < argc; ++i) {
        std::string arg = argv[i];
        if (arg == "--num_blocks" && i + 1 < argc) num_blocks = std::atoi(argv[++i]);
        else if (arg == "--num_threads" && i + 1 < argc) num_threads = std::atoi(argv[++i]);
        else if (arg == "--num_calls" && i + 1 < argc) num_calls = std::atoi(argv[++i]);
        else if (arg == "--kernel" && i + 1 < argc) kernel_number = std::atoi(argv[++i]);
        else if (arg == "--verbose") enable_logging = true;
        else if (arg == "--random") randomize = true;
    }
    
    std::vector<KernelConfig> configs;
    for (int i = 0; i < MAX_KERNELS; ++i) {
        configs.push_back({
            randomize ? dist_blocks(gen) : num_blocks,
            randomize ? dist_threads(gen) : num_threads,
            randomize ? dist_calls(gen) : num_calls,
            randomize ? dist_kernel(gen) : kernel_number
        });
    }
    
    run_kernel(rank, configs, enable_logging);
    
    MPI_Finalize();
    return 0;
}