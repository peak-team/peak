#include <hip/hip_runtime.h>
#include <pthread.h>
#include <iostream>
#include <cstdlib>
#include <sstream>
#include <atomic>
#include <vector>
#include <random>

#define LINE_WIDTH 80
#define MAX_KERNELS 5

// Define up to 5 kernels with a logging flag
__global__ void kernel1(unsigned long thread_id, int call_id, bool enable_logging) {
    if (enable_logging) {
        printf("[KERNEL 1] GPU Block: %d | GPU Thread: %d | Call: %d | Pthread ID: %lu\n",
               blockIdx.x, threadIdx.x, call_id, thread_id);
    }
}

__global__ void kernel2(unsigned long thread_id, int call_id, bool enable_logging) {
    if (enable_logging) {
        printf("[KERNEL 2] GPU Block: %d | GPU Thread: %d | Call: %d | Pthread ID: %lu\n",
               blockIdx.x, threadIdx.x, call_id, thread_id);
    }
}

__global__ void kernel3(unsigned long thread_id, int call_id, bool enable_logging) {
    if (enable_logging) {
        printf("[KERNEL 3] GPU Block: %d | GPU Thread: %d | Call: %d | Pthread ID: %lu\n",
               blockIdx.x, threadIdx.x, call_id, thread_id);
    }
}

__global__ void kernel4(unsigned long thread_id, int call_id, bool enable_logging) {
    if (enable_logging) {
        printf("[KERNEL 4] GPU Block: %d | GPU Thread: %d | Call: %d | Pthread ID: %lu\n",
               blockIdx.x, threadIdx.x, call_id, thread_id);
    }
}

__global__ void kernel5(unsigned long thread_id, int call_id, bool enable_logging) {
    if (enable_logging) {
        printf("[KERNEL 5] GPU Block: %d | GPU Thread: %d | Call: %d | Pthread ID: %lu\n",
               blockIdx.x, threadIdx.x, call_id, thread_id);
    }
}

struct ThreadArgs {
    int gpu_id;
    std::vector<int> num_blocks;
    std::vector<int> num_threads;
    std::vector<int> num_calls;
    std::vector<int> kernel_numbers;
    bool enable_logging;
};

std::atomic<int> total_calls(0);
std::atomic<int> total_blocks(0);
std::atomic<int> total_threads(0);

void print_line() {
    for (int i = 0; i < LINE_WIDTH; ++i) std::cout << "=";
    std::cout << "\n";
}

void* run_kernel(void* arg) {
    ThreadArgs* args = (ThreadArgs*)arg;
    hipSetDevice(args->gpu_id);
    pthread_t thread_id = pthread_self();
    
    if (args->enable_logging) {
        print_line();
        std::cout << "Thread " << thread_id << " running on GPU " << args->gpu_id << "\n";
        print_line();
        for (size_t i = 0; i < args->kernel_numbers.size(); ++i) {
            std::cout << "Kernel " << args->kernel_numbers[i] << " | Blocks: " << args->num_blocks[i]
                      << " | Threads: " << args->num_threads[i] << " | Calls: " << args->num_calls[i] << "\n";
        }
        print_line();
    }
    
    for (size_t i = 0; i < args->kernel_numbers.size(); ++i) {
        int blocks = args->num_blocks[i];
        int threads = args->num_threads[i];
        int calls = args->num_calls[i];
        int kernel = args->kernel_numbers[i];
        
        for (int j = 0; j < calls; ++j) {
            switch (kernel) {
                case 1:
                    kernel1<<<blocks, threads>>>((unsigned long)thread_id, j, args->enable_logging);
                    break;
                case 2:
                    kernel2<<<blocks, threads>>>((unsigned long)thread_id, j, args->enable_logging);
                    break;
                case 3:
                    kernel3<<<blocks, threads>>>((unsigned long)thread_id, j, args->enable_logging);
                    break;
                case 4:
                    kernel4<<<blocks, threads>>>((unsigned long)thread_id, j, args->enable_logging);
                    break;
                case 5:
                    kernel5<<<blocks, threads>>>((unsigned long)thread_id, j, args->enable_logging);
                    break;
                default:
                    return nullptr;
            }
            hipDeviceSynchronize();
            total_calls++;
            total_blocks += blocks;
            total_threads += threads;
        }
    }
    return nullptr;
}

int main(int argc, char** argv) {
    int num_cpu_threads = 1;
    int num_blocks = 1;
    int num_threads = 32;
    int num_calls = 1;
    int kernel_number = 1;
    bool enable_logging = false;
    bool randomize = false;
    
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<int> dist_blocks(1, 16);
    std::uniform_int_distribution<int> dist_threads(32, 512);
    std::uniform_int_distribution<int> dist_calls(1, 10);
    std::uniform_int_distribution<int> dist_kernel(1, 5);
    
    for (int i = 1; i < argc; ++i) {
        std::string arg = argv[i];
        if (arg == "--num_pthreads" && i + 1 < argc) num_cpu_threads = std::atoi(argv[++i]);
        else if (arg == "--num_blocks" && i + 1 < argc) num_blocks = std::atoi(argv[++i]);
        else if (arg == "--num_threads" && i + 1 < argc) num_threads = std::atoi(argv[++i]);
        else if (arg == "--num_calls" && i + 1 < argc) num_calls = std::atoi(argv[++i]);
        else if (arg == "--kernel" && i + 1 < argc) kernel_number = std::atoi(argv[++i]);
        else if (arg == "--verbose") enable_logging = true;
        else if (arg == "--random") randomize = true;
    }
    
    pthread_t threads[num_cpu_threads];
    ThreadArgs thread_args[num_cpu_threads];
    
    for (int t = 0; t < num_cpu_threads; ++t) {
        for (int i = 0; i < MAX_KERNELS; ++i) {
            thread_args[t].num_blocks.push_back(randomize ? dist_blocks(gen) : num_blocks);
            thread_args[t].num_threads.push_back(randomize ? dist_threads(gen) : num_threads);
            thread_args[t].num_calls.push_back(randomize ? dist_calls(gen) : num_calls);
            thread_args[t].kernel_numbers.push_back(randomize ? dist_kernel(gen) : kernel_number);
        }
        thread_args[t].gpu_id = 0;
        thread_args[t].enable_logging = enable_logging;
        pthread_create(&threads[t], nullptr, run_kernel, (void*)&thread_args[t]);
    }
    
    for (int t = 0; t < num_cpu_threads; ++t) {
        pthread_join(threads[t], nullptr);
    }
    
    if (enable_logging) {
        print_line();
        std::cout << "Execution completed!\n";
        print_line();
    }

    double avg_blocks = static_cast<double>(total_blocks) / total_calls;
    double avg_threads = static_cast<double>(total_threads) / total_calls;

    std::cout << "Total Kernel Calls: " << total_calls.load() << "\n";
    std::cout << "Average Block Size: " << avg_blocks << "\n";
    std::cout << "Average Grid Size: " << avg_threads << "\n";
    
    return 0;
}